#include "hip/hip_runtime.h"
extern "C" {
	#include "lua.h"
	#include "lualib.h"
	#include "lauxlib.h"
}

#include "luaT.h"
#include "THC.h"

#include <stdio.h>
#include <assert.h>
#include <hip/hip_math_constants.h>
#include <stdint.h>
#include <unistd.h>
#include <png++/image.hpp>

#define TB 128

#define DISP_MAX 256

THCState* getCutorchState(lua_State* L)
{
	lua_getglobal(L, "cutorch");
	lua_getfield(L, -1, "getState");
	lua_call(L, 0, 1);
	THCState *state = (THCState*) lua_touserdata(L, -1);
	lua_pop(L, 2);
	return state;
}

void checkCudaError(lua_State *L) {
	hipError_t status = hipPeekAtLastError();
	if (status != hipSuccess) {
		luaL_error(L, hipGetErrorString(status));
	}
}

#define COLOR_DIFF(x, i, j) (abs(x[i] - x[j]))

THCudaTensor *new_tensor_like(THCState *state, THCudaTensor *x)
{
	THCudaTensor *y = THCudaTensor_new(state);
	THCudaTensor_resizeAs(state, y, x);
	return y;
}

__device__ void sort(float *x, int n)
{
	for (int i = 0; i < n - 1; i++) {
		int min = i;
		for (int j = i + 1; j < n; j++) {
			if (x[j] < x[min]) {
				min = j;
			}
		}
		float tmp = x[min];
		x[min] = x[i];
		x[i] = tmp;
	}
}

__global__ void ad(float *x0, float *x1, float *output, int size, int size2, int size3, int direction)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < size) {
		int d = id;
		int x = d % size3;
		d /= size3;
		int y = d % size2;
		d /= size2;
		d *= direction;

		float dist;
		if (0 <= x + d && x + d < size3) {
			int cnt = 0;
			dist = 0;
			for (int yy = y - 4; yy <= y + 4; yy++) {
				for (int xx = x - 4; xx <= x + 4; xx++) {
					if (0 <= xx && xx < size3 && 0 <= xx + d && xx + d < size3 && 0 <= yy && yy < size2) {
						int ind = yy * size3 + xx;
						dist += abs(x0[ind] - x1[ind + d]);
						cnt++;
					}
				}
			}
			dist /= cnt;
		} else {
			dist = HIP_NAN;
		}
		output[id] = dist;
	}
}

int ad(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *x0 = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *x1 = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *out = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	int direction = luaL_checkinteger(L, 4);
	assert(direction == -1 || direction == 1);

	ad<<<(THCudaTensor_nElement(state, out) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, x0),
		THCudaTensor_data(state, x1),
		THCudaTensor_data(state, out),
		THCudaTensor_nElement(state, out),
		THCudaTensor_size(state, out, 2),
		THCudaTensor_size(state, out, 3),
		direction);
	checkCudaError(L);
	return 0;
}


__global__ void census(float *x0, float *x1, float *output, int size, int num_channels, int size2, int size3, int direction)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < size) {
		int d = id;
		int x = d % size3;
		d /= size3;
		int y = d % size2;
		d /= size2;
		d *= direction;

		float dist;
		if (0 <= x + d && x + d < size3) {
			dist = 0;
			for (int i = 0; i < num_channels; i++) {
				int ind_p = (i * size2 + y) * size3 + x;
				for (int yy = y - 4; yy <= y + 4; yy++) {
					for (int xx = x - 4; xx <= x + 4; xx++) {
						if (0 <= xx && xx < size3 && 0 <= xx + d && xx + d < size3 && 0 <= yy && yy < size2) {
							int ind_q = (i * size2 + yy) * size3 + xx;
							if ((x0[ind_q] < x0[ind_p]) != (x1[ind_q + d] < x1[ind_p + d])) {
								dist++;
							}
						} else {
							dist++;
						}
					}
				}
			}
			dist /= num_channels;
		} else {
			dist = HIP_NAN;
		}
		output[id] = dist;
	}
}

int census(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *x0 = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *x1 = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *out = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	int direction = luaL_checkinteger(L, 4);
	assert(direction == -1 || direction == 1);

	census<<<(THCudaTensor_nElement(state, out) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, x0),
		THCudaTensor_data(state, x1),
		THCudaTensor_data(state, out),
		THCudaTensor_nElement(state, out),
		THCudaTensor_size(state, x0, 1),
		THCudaTensor_size(state, out, 2),
		THCudaTensor_size(state, out, 3),
		direction);
	checkCudaError(L);
	return 0;
}

#if 0
__global__ void add_vol(float *vol, float *cnt, float *out, int size, int size1, int size2, int size3, float ratio)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int d = id;
		int x = d % size3;
		d /= size3;
		int y = d % size2;
		d /= size2;

		int lo = floor(d * ratio);
		int hi = lo + 1;
		float alpha = (d * ratio) - lo;
		assert(0 <= lo && hi < size1);

		float val = vol[(lo * size2 + y) * size3 + x] * (1 - alpha) + vol[(hi * size2 + y) * size3 + x] * alpha;
		if (!isnan(val) && cnt[id] > 0) {
			out[id] += val;
			cnt[id] += 1;
		}
	}
}

int add_vol(lua_State *L)
{
	THCudaTensor *vol = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *cnt = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *out = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	float ratio = luaL_checknumber(L, 4);

	add_vol<<<(THCudaTensor_nElement(out) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(vol),
		THCudaTensor_data(cnt),
		THCudaTensor_data(out),
		THCudaTensor_nElement(out),
		THCudaTensor_size(vol, 1),
		THCudaTensor_size(out, 2),
		THCudaTensor_size(out, 3),
		ratio);
	checkCudaError(L);
	return 0;
}

__global__ void rho(float *x, int size, float lambda)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		x[id] = 1 - exp(-x[id] / lambda);
	}
}

int rho(lua_State *L)
{
	THCudaTensor *x = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	float lambda = luaL_checknumber(L, 2);

	rho<<<(THCudaTensor_nElement(x) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(x), 
		THCudaTensor_nElement(x),
		lambda);
	checkCudaError(L);
	return 0;
}

#endif

__global__ void spatial_argmin(float *input, float *output, int size, int size1, int size23)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int dim23 = id % size23;
		int dim0 = id / size23;

		int argmin = 0;
		float min = HIP_INF;
		for (int i = 0; i < size1; i++) {
			float val = input[(dim0 * size1 + i) * size23 + dim23];
			if (val < min) {
				min = val;
				argmin = i;
			}
		}
		output[id] = argmin + 1;
	}
}

int spatial_argmin(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *output = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");

	spatial_argmin<<<(THCudaTensor_nElement(state, output) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, input),
		THCudaTensor_data(state, output),
		THCudaTensor_nElement(state, output),
		THCudaTensor_size(state, input, 1),
		THCudaTensor_size(state, input, 2) * THCudaTensor_size(state, output, 3));
	checkCudaError(L);
	return 0;
}

__global__ void cross(float *x0, float *out, int size, int dim2, int dim3, int L1, float tau1)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int dir = id;
		int x = dir % dim3;
		dir /= dim3;
		int y = dir % dim2;
		dir /= dim2;

		int dx = 0;
		int dy = 0;
		if (dir == 0) {
			dx = -1;
		} else if (dir == 1) {
			dx = 1;
		} else if (dir == 2) {
			dy = -1;
		} else if (dir == 3) {
			dy = 1;
		} else {
			assert(0);
		}

		int xx, yy, ind1, ind2, dist;
		ind1 = y * dim3 + x;
		for (xx = x + dx, yy = y + dy;;xx += dx, yy += dy) {
			if (xx < 0 || xx >= dim3 || yy < 0 || yy >= dim2) break;

			dist = max(abs(xx - x), abs(yy - y));
			if (dist == 1) continue;

			ind2 = yy * dim3 + xx;

			/* rule 1 */
			if (COLOR_DIFF(x0, ind1, ind2) >= tau1) break;

			/* rule 2 */
			if (dist >= L1) break;
		}
		out[id] = dir <= 1 ? xx : yy;
	}
}

int cross(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *x0 = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *out = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	int L1 = luaL_checkinteger(L, 3);
	float tau1 = luaL_checknumber(L, 4);

	cross<<<(THCudaTensor_nElement(state, out) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, x0),
		THCudaTensor_data(state, out),
		THCudaTensor_nElement(state, out),
		THCudaTensor_size(state, out, 2),
		THCudaTensor_size(state, out, 3),
		L1, tau1);
	checkCudaError(L);
	return 0;
}

__global__ void cbca(float *x0c, float *x1c, float *vol, float *out, int size, int dim2, int dim3, int direction)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int d = id;
		int x = d % dim3;
		d /= dim3;
		int y = d % dim2;
		d /= dim2;

		if (x + d * direction < 0 || x + d * direction >= dim3) {
			out[id] = vol[id];
		} else {
			float sum = 0;
			int cnt = 0;

			int yy_s = max(x0c[(2 * dim2 + y) * dim3 + x], x1c[(2 * dim2 + y) * dim3 + x + d * direction]);
			int yy_t = min(x0c[(3 * dim2 + y) * dim3 + x], x1c[(3 * dim2 + y) * dim3 + x + d * direction]);
			for (int yy = yy_s + 1; yy < yy_t; yy++) {
				int xx_s = max(x0c[(0 * dim2 + yy) * dim3 + x], x1c[(0 * dim2 + yy) * dim3 + x + d * direction] - d * direction);
				int xx_t = min(x0c[(1 * dim2 + yy) * dim3 + x], x1c[(1 * dim2 + yy) * dim3 + x + d * direction] - d * direction);
				for (int xx = xx_s + 1; xx < xx_t; xx++) {
					float val = vol[(d * dim2 + yy) * dim3 + xx];
					assert(!isnan(val));
					sum += val;
					cnt++;
				}
			}

			assert(cnt > 0);
			out[id] = sum / cnt;
			assert(!isnan(out[id]));
		}
	}
}

int cbca(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *x0c = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *x1c = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *vol_in = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	THCudaTensor *vol_out = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
	int direction = luaL_checkinteger(L, 5);

	assert(direction == -1 or direction == 1);
	cbca<<<(THCudaTensor_nElement(state, vol_out) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, x0c),
		THCudaTensor_data(state, x1c),
		THCudaTensor_data(state, vol_in),
		THCudaTensor_data(state, vol_out),
		THCudaTensor_nElement(state, vol_out),
		THCudaTensor_size(state, vol_out, 2),
		THCudaTensor_size(state, vol_out, 3),
		direction);
	checkCudaError(L);
	return 0;
}

__global__ void sgm(float *x0, float *x1, float *vol, float *tmp, float *out, int dim1, int dim2, int dim3, float pi1, float pi2, float tau_so, float alpha1, float sgm_q1, float sgm_q2, int sgm_direction, int direction)
{
	int x, y, dx, dy;

	dx = dy = 0;
	if (sgm_direction <= 1) {
		y = blockIdx.x * blockDim.x + threadIdx.x;
		if (y >= dim2) {
			return;
		}
		if (sgm_direction == 0) {
			x = 0;
			dx = 1;
		} else if (sgm_direction == 1) {
			x = dim3 - 1;
			dx = -1;
		}
	} else if (sgm_direction <= 3) {
		x = blockIdx.x * blockDim.x + threadIdx.x;
		if (x >= dim3) {
			return;
		}
		if (sgm_direction == 2) {
			y = 0;
			dy = 1;
		} else if (sgm_direction == 3) {
			y = dim2 - 1;
			dy = -1;
		}
	}

	assert(dim1 <= 400);
	float tmp_curr_[400];
	float tmp_prev_[400];
	float *tmp_curr = tmp_curr_;
	float *tmp_prev = tmp_prev_;

	float min_prev = HIP_INF;
	for (; 0 <= y && y < dim2 && 0 <= x && x < dim3; x += dx, y += dy) {
		float min_curr = HIP_INF;
		for (int d = 0; d < dim1; d++) {
			int ind = (d * dim2 + y) * dim3 + x;

			if (x + d * direction < 0 ||
				x + d * direction >= dim3 || 
				y - dy < 0 || 
				y - dy >= dim2 || 
				x + d * direction - dx < 0 || 
				x + d * direction - dx >= dim3 ||
				x - dx < 0 ||
				x - dx >= dim3) {

				out[ind] += vol[ind];
				tmp_curr[d] = vol[ind];
			} else {
				int ind2 = y * dim3 + x;

				float D1 = COLOR_DIFF(x0, ind2, ind2 - dy * dim3 - dx);
				float D2 = COLOR_DIFF(x1, ind2 + d * direction, ind2 + d * direction - dy * dim3 - dx);
				float P1, P2;
				if (D1 < tau_so && D2 < tau_so) { 
					P1 = pi1; 
					P2 = (pi1 * pi2); 
				} else if (D1 > tau_so && D2 > tau_so) { 
					P1 = pi1 / (sgm_q1 * sgm_q2);
					P2 = (pi1 * pi2) / (sgm_q1 * sgm_q2);
				} else {
					P1 = pi1 / sgm_q1;
					P2 = (pi1 * pi2) / sgm_q1;
				}

				assert(min_prev != HIP_INF);
				float cost = min(tmp_prev[d], min_prev + P2);
				if (d > 0) {
					cost = min(cost, tmp_prev[d - 1] + (sgm_direction == 2 ? P1 / alpha1 : P1));
				}
				if (d < dim1 - 1) {
					cost = min(cost, tmp_prev[d + 1] + (sgm_direction == 3 ? P1 / alpha1 : P1));
				}
				float val = vol[ind] + cost - min_prev;
				out[ind] += val;
				tmp_curr[d] = val;
			}
			if (tmp_curr[d] < min_curr) {
				min_curr = tmp_curr[d];
			}
		}
		min_prev = min_curr;

		float *swap = tmp_curr;
		tmp_curr = tmp_prev;
		tmp_prev = swap;
	}
}

int sgm(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *x0 = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *x1 = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *vol = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	THCudaTensor *tmp = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
	THCudaTensor *out = (THCudaTensor*)luaT_checkudata(L, 5, "torch.CudaTensor");
	float pi1 = luaL_checknumber(L, 6);
	float pi2 = luaL_checknumber(L, 7);
	float tau_so = luaL_checknumber(L, 8);
	float alpha1 = luaL_checknumber(L, 9);
	float sgm_q1 = luaL_checknumber(L, 10);
	float sgm_q2 = luaL_checknumber(L, 11);
	int direction = luaL_checknumber(L, 12);

	int dim1 = THCudaTensor_size(state, out, 1);
	int dim2 = THCudaTensor_size(state, out, 2);
	int dim3 = THCudaTensor_size(state, out, 3);

	for (int sgm_direction = 0; sgm_direction < 4; sgm_direction++) {
		int size = sgm_direction <= 1 ? dim2 : dim3;
		sgm<<<(size - 1) / TB + 1, TB>>>(
			THCudaTensor_data(state, x0),
			THCudaTensor_data(state, x1),
			THCudaTensor_data(state, vol),
			THCudaTensor_data(state, tmp),
			THCudaTensor_data(state, out),
			dim1, dim2, dim3, pi1, pi2, tau_so, alpha1, sgm_q1, sgm_q2, sgm_direction, direction);
	}
	checkCudaError(L);
	return 0;
}

#define INDEX(dim0, dim1, dim2, dim3) \
	assert((dim1) >= 0 && (dim1) < size1 && (dim2) >= 0 && (dim2) < size2 && (dim3) >= 0 && (dim3) < size3), \
	((((dim0) * size1 + (dim1)) * size2 + (dim2)) * size3 + dim3)

template <int sgm_direction>
__global__ void sgm2(float *x0, float *x1, float *input, float *output, float *tmp, float pi1, float pi2, float tau_so, float alpha1, float sgm_q1, float sgm_q2, int direction, int size1, int size2, int size3, int step)
{
	int x, y, dx, dy;
	int d = threadIdx.x;

	if (sgm_direction == 0) {
		/* right */
		x = step;
		y = blockIdx.x;
		dx = 1;
		dy = 0;
	} else if (sgm_direction == 1) {
		/* left */
		x = size2 - 1 - step;
		y = blockIdx.x;
		dx = -1;
		dy = 0;
	} else if (sgm_direction == 2) {
		/* down */
		x = blockIdx.x;
		y = step;
		dx = 0;
		dy = 1;
	} else if (sgm_direction == 3) {
		/* up */
		x = blockIdx.x;
		y = size1 - 1 - step;
		dx = 0;
		dy = -1;
	}

	if (y - dy < 0 || y - dy >= size1 || x - dx < 0 || x - dx >= size2) {
		float val = input[INDEX(0, y, x, d)];
		output[INDEX(0, y, x, d)] += val;
		tmp[d * size2 + blockIdx.x] = val;
		return;
	}

	__shared__ float output_s[400], output_min[400];

	output_s[d] = output_min[d] = tmp[d * size2 + blockIdx.x];
	__syncthreads();

	for (int i = 256; i > 0; i /= 2) {
		if (d < i && d + i < size3 && output_min[d + i] < output_min[d]) {
			output_min[d] = output_min[d + i];
		}
		__syncthreads();
	}

	int ind2 = y * size2 + x;
	float D1 = COLOR_DIFF(x0, ind2, ind2 - dy * size2 - dx);
	float D2;
	int xx = x + d * direction;
	if (xx < 0 || xx >= size2 || xx - dx < 0 || xx - dx >= size2) {
		D2 = 10;
	} else {
		D2 = COLOR_DIFF(x1, ind2 + d * direction, ind2 + d * direction - dy * size2 - dx);
	}
	float P1, P2;
	if (D1 < tau_so && D2 < tau_so) {
		P1 = pi1;
		P2 = pi2;
	} else if (D1 > tau_so && D2 > tau_so) {
		P1 = pi1 / (sgm_q1 * sgm_q2);
		P2 = pi2 / (sgm_q1 * sgm_q2);
	} else {
		P1 = pi1 / sgm_q1;
		P2 = pi2 / sgm_q1;
	}

	float cost = min(output_s[d], output_min[0] + P2);
	if (d - 1 >= 0) {
		cost = min(cost, output_s[d - 1] + (sgm_direction == 2 ? P1 / alpha1 : P1));
	}
	if (d + 1 < size3) {
		cost = min(cost, output_s[d + 1] + (sgm_direction == 3 ? P1 / alpha1 : P1));
	}

	float val = input[INDEX(0, y, x, d)] + cost - output_min[0];
	output[INDEX(0, y, x, d)] += val;
	tmp[d * size2 + blockIdx.x] = val;
}

int sgm2(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *x0 = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *x1 = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	THCudaTensor *output = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
	THCudaTensor *tmp = (THCudaTensor*)luaT_checkudata(L, 5, "torch.CudaTensor");
	float pi1 = luaL_checknumber(L, 6);
	float pi2 = luaL_checknumber(L, 7);
	float tau_so = luaL_checknumber(L, 8);
	float alpha1 = luaL_checknumber(L, 9);
	float sgm_q1 = luaL_checknumber(L, 10);
	float sgm_q2 = luaL_checknumber(L, 11);
	int direction = luaL_checknumber(L, 12);
	int size1 = THCudaTensor_size(state, output, 1) * THCudaTensor_size(state, output, 3);
	int size2 = THCudaTensor_size(state, output, 2) * THCudaTensor_size(state, output, 3);
	int disp_max = THCudaTensor_size(state, output, 3);

	for (int step = 0; step < THCudaTensor_size(state, input, 2); step++) {
		sgm2<0><<<(size1 - 1) / disp_max + 1, disp_max>>>(
			THCudaTensor_data(state, x0),
			THCudaTensor_data(state, x1),
			THCudaTensor_data(state, input),
			THCudaTensor_data(state, output),
			THCudaTensor_data(state, tmp),
			pi1, pi2, tau_so, alpha1, sgm_q1, sgm_q2, direction,
			THCudaTensor_size(state, input, 1),
			THCudaTensor_size(state, input, 2),
			THCudaTensor_size(state, input, 3),
			step);
	}

	for (int step = 0; step < THCudaTensor_size(state, input, 2); step++) {
		sgm2<1><<<(size1 - 1) / disp_max + 1, disp_max>>>(
			THCudaTensor_data(state, x0),
			THCudaTensor_data(state, x1),
			THCudaTensor_data(state, input),
			THCudaTensor_data(state, output),
			THCudaTensor_data(state, tmp),
			pi1, pi2, tau_so, alpha1, sgm_q1, sgm_q2, direction,
			THCudaTensor_size(state, input, 1),
			THCudaTensor_size(state, input, 2),
			THCudaTensor_size(state, input, 3),
			step);
	}

	for (int step = 0; step < THCudaTensor_size(state, input, 1); step++) {
		sgm2<2><<<(size2 - 1) / disp_max + 1, disp_max>>>(
			THCudaTensor_data(state, x0),
			THCudaTensor_data(state, x1),
			THCudaTensor_data(state, input),
			THCudaTensor_data(state, output),
			THCudaTensor_data(state, tmp),
			pi1, pi2, tau_so, alpha1, sgm_q1, sgm_q2, direction,
			THCudaTensor_size(state, input, 1),
			THCudaTensor_size(state, input, 2),
			THCudaTensor_size(state, input, 3),
			step);
	}

	for (int step = 0; step < THCudaTensor_size(state, input, 1); step++) {
		sgm2<3><<<(size2 - 1) / disp_max + 1, disp_max>>>(
			THCudaTensor_data(state, x0),
			THCudaTensor_data(state, x1),
			THCudaTensor_data(state, input),
			THCudaTensor_data(state, output),
			THCudaTensor_data(state, tmp),
			pi1, pi2, tau_so, alpha1, sgm_q1, sgm_q2, direction,
			THCudaTensor_size(state, input, 1),
			THCudaTensor_size(state, input, 2),
			THCudaTensor_size(state, input, 3),
			step);
	}

	checkCudaError(L);
	return 0;
}

template <int sgm_direction>
__global__ void sgm3(float *x0, float *x1, float *input, float *output, float pi1, float pi2, float tau_so, float alpha1, float sgm_q1, float sgm_q2, int direction, int size1, int size2, int size3, int step)
{
	int x, y, dx, dy;
	int d = threadIdx.x;

	if (sgm_direction == 0) {
		/* right */
		x = step;
		y = blockIdx.x;
		dx = 1;
		dy = 0;
	} else if (sgm_direction == 1) {
		/* left */
		x = size2 - 1 - step;
		y = blockIdx.x;
		dx = -1;
		dy = 0;
	} else if (sgm_direction == 2) {
		/* down */
		x = blockIdx.x;
		y = step;
		dx = 0;
		dy = 1;
	} else if (sgm_direction == 3) {
		/* up */
		x = blockIdx.x;
		y = size1 - 1 - step;
		dx = 0;
		dy = -1;
	}

	if (y - dy < 0 || y - dy >= size1 || x - dx < 0 || x - dx >= size2) {
		output[INDEX(sgm_direction, y, x, d)] = input[INDEX(0, y, x, d)];
		return;
	}

	__shared__ float output_s[400], output_min[400];

	output_s[d] = output_min[d] = output[INDEX(sgm_direction, y - dy, x - dx, d)];
	__syncthreads();

	for (int i = 256; i > 0; i /= 2) {
		if (d < i && d + i < size3 && output_min[d + i] < output_min[d]) {
			output_min[d] = output_min[d + i];
		}
		__syncthreads();
	}

	int ind2 = y * size2 + x;
	float D1 = COLOR_DIFF(x0, ind2, ind2 - dy * size2 - dx);
	float D2;
	int xx = x + d * direction;
	if (xx < 0 || xx >= size2 || xx - dx < 0 || xx - dx >= size2) {
		D2 = 10;
	} else {
		D2 = COLOR_DIFF(x1, ind2 + d * direction, ind2 + d * direction - dy * size2 - dx);
	}
	float P1, P2;
	if (D1 < tau_so && D2 < tau_so) {
		P1 = pi1;
		P2 = pi2;
	} else if (D1 > tau_so && D2 > tau_so) {
		P1 = pi1 / (sgm_q1 * sgm_q2);
		P2 = pi2 / (sgm_q1 * sgm_q2);
	} else {
		P1 = pi1 / sgm_q1;
		P2 = pi2 / sgm_q1;
	}

	float cost = min(output_s[d], output_min[0] + P2);
	if (d - 1 >= 0) {
		cost = min(cost, output_s[d - 1] + (sgm_direction == 2 ? P1 / alpha1 : P1));
	}
	if (d + 1 < size3) {
		cost = min(cost, output_s[d + 1] + (sgm_direction == 3 ? P1 / alpha1 : P1));
	}

	output[INDEX(sgm_direction, y, x, d)] = input[INDEX(0, y, x, d)] + cost - output_min[0];
}

int sgm3(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *x0 = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *x1 = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	THCudaTensor *output = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
	float pi1 = luaL_checknumber(L, 5);
	float pi2 = luaL_checknumber(L, 6);
	float tau_so = luaL_checknumber(L, 7);
	float alpha1 = luaL_checknumber(L, 8);
	float sgm_q1 = luaL_checknumber(L, 9);
	float sgm_q2 = luaL_checknumber(L, 10);
	int direction = luaL_checknumber(L, 11);
	int size1 = THCudaTensor_size(state, output, 1) * THCudaTensor_size(state, output, 3);
	int size2 = THCudaTensor_size(state, output, 2) * THCudaTensor_size(state, output, 3);
	int disp_max = THCudaTensor_size(state, output, 3);

	for (int step = 0; step < THCudaTensor_size(state, input, 2); step++) {
		sgm3<0><<<(size1 - 1) / disp_max + 1, disp_max>>>(
			THCudaTensor_data(state, x0),
			THCudaTensor_data(state, x1),
			THCudaTensor_data(state, input),
			THCudaTensor_data(state, output),
			pi1, pi2, tau_so, alpha1, sgm_q1, sgm_q2, direction,
			THCudaTensor_size(state, input, 1),
			THCudaTensor_size(state, input, 2),
			THCudaTensor_size(state, input, 3),
			step);
	}

	for (int step = 0; step < THCudaTensor_size(state, input, 2); step++) {
		sgm3<1><<<(size1 - 1) / disp_max + 1, disp_max>>>(
			THCudaTensor_data(state, x0),
			THCudaTensor_data(state, x1),
			THCudaTensor_data(state, input),
			THCudaTensor_data(state, output),
			pi1, pi2, tau_so, alpha1, sgm_q1, sgm_q2, direction,
			THCudaTensor_size(state, input, 1),
			THCudaTensor_size(state, input, 2),
			THCudaTensor_size(state, input, 3),
			step);
	}

	for (int step = 0; step < THCudaTensor_size(state, input, 1); step++) {
		sgm3<2><<<(size2 - 1) / disp_max + 1, disp_max>>>(
			THCudaTensor_data(state, x0),
			THCudaTensor_data(state, x1),
			THCudaTensor_data(state, input),
			THCudaTensor_data(state, output),
			pi1, pi2, tau_so, alpha1, sgm_q1, sgm_q2, direction,
			THCudaTensor_size(state, input, 1),
			THCudaTensor_size(state, input, 2),
			THCudaTensor_size(state, input, 3),
			step);
	}

	for (int step = 0; step < THCudaTensor_size(state, input, 1); step++) {
		sgm3<3><<<(size2 - 1) / disp_max + 1, disp_max>>>(
			THCudaTensor_data(state, x0),
			THCudaTensor_data(state, x1),
			THCudaTensor_data(state, input),
			THCudaTensor_data(state, output),
			pi1, pi2, tau_so, alpha1, sgm_q1, sgm_q2, direction,
			THCudaTensor_size(state, input, 1),
			THCudaTensor_size(state, input, 2),
			THCudaTensor_size(state, input, 3),
			step);
	}

	checkCudaError(L);
	return 0;
}

__global__ void fliplr(float *in, float *out, int size, int dim3)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int x = id % dim3;
		out[id + dim3 - 2 * x - 1] = in[id];
	}
}

int fliplr(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *in = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *out = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");

	fliplr<<<(THCudaTensor_nElement(state, out) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, in),
		THCudaTensor_data(state, out),
		THCudaTensor_nElement(state, out),
		THCudaTensor_size(state, out, 3));
	checkCudaError(L);
	return 0;
}

__global__ void outlier_detection(float *d0, float *d1, float *outlier, int size, int dim3, int disp_max)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int x = id % dim3;
		int d0i = d0[id];
		if (x - d0i < 0) {
			//assert(0);
			outlier[id] = 1;
		} else if (abs(d0[id] - d1[id - d0i]) < 1.1) {
			outlier[id] = 0; /* match */
		} else {
			outlier[id] = 1; /* occlusion */
			for (int d = 0; d < disp_max; d++) {
				if (x - d >= 0 && abs(d - d1[id - d]) < 1.1) {
					outlier[id] = 2; /* mismatch */
					break;
				}
			}
		}
	}
}

int outlier_detection(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *d0 = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *d1 = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *outlier = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	int disp_max = luaL_checkinteger(L, 4);

	outlier_detection<<<(THCudaTensor_nElement(state, d0) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, d0),
		THCudaTensor_data(state, d1),
		THCudaTensor_data(state, outlier),
		THCudaTensor_nElement(state, d0),
		THCudaTensor_size(state, d0, 3),
		disp_max);
	checkCudaError(L);
	return 0;
}

#if 0

__global__ void iterative_region_voting(float *d0, float *x0c, float *x1c, float *outlier, float *d0_out, float *outlier_out, int size, int dim2, int dim3, float tau_s, float tau_h, int disp_max)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int x = id % dim3;
		int y = id / dim3;
		
		d0_out[id] = d0[id];
		outlier_out[id] = outlier[id];

		if (outlier[id] == 0) return;

		assert(disp_max < DISP_MAX);
		int hist[DISP_MAX];
		for (int i = 0; i < disp_max; i++) {
			hist[i] = 0;
		}

		int yy_s = x0c[(2 * dim2 + y) * dim3 + x];
		int yy_t = x0c[(3 * dim2 + y) * dim3 + x];
		for (int yy = yy_s + 1; yy < yy_t; yy++) {
			int xx_s = x0c[(0 * dim2 + yy) * dim3 + x];
			int xx_t = x0c[(1 * dim2 + yy) * dim3 + x];
			for (int xx = xx_s + 1; xx < xx_t; xx++) {
				if (outlier[yy * dim3 + xx] == 0) {
					hist[(int)d0[yy * dim3 + xx]]++;
				}
			}
		}

		int cnt = 0;
		int max_i = 0;
		for (int i = 0; i < disp_max; i++) {
			cnt += hist[i];
			if (hist[i] > hist[max_i]) {
				max_i = i;
			}
		}

		if (cnt > tau_s && (float)hist[max_i] / cnt > tau_h) {
			outlier_out[id] = 0;
			d0_out[id] = max_i;
		}
	}
}

int iterative_region_voting(lua_State *L)
{
	THCudaTensor *d0 = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *x0c = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *x1c = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	THCudaTensor *outlier = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
	float tau_s = luaL_checknumber(L, 5);
	float tau_h = luaL_checknumber(L, 6);
	int disp_max = luaL_checkinteger(L, 7);
	int iterations = luaL_checkinteger(L, 8);

	THCudaTensor *d0_tmp = new_tensor_like(state, d0);
	THCudaTensor *outlier_tmp = new_tensor_like(state, outlier);

	assert(iterations % 2 == 0);
	for (int i = 0; i < iterations; i++) {
		iterative_region_voting<<<(THCudaTensor_nElement(d0) - 1) / TB + 1, TB>>>(
			THCudaTensor_data(i % 2 == 0 ? d0 : d0_tmp),
			THCudaTensor_data(x0c),
			THCudaTensor_data(x1c),
			THCudaTensor_data(i % 2 == 0 ? outlier : outlier_tmp),
			THCudaTensor_data(i % 2 == 0 ? d0_tmp : d0),
			THCudaTensor_data(i % 2 == 0 ? outlier_tmp : outlier),
			THCudaTensor_nElement(d0),
			THCudaTensor_size(d0, 2),
			THCudaTensor_size(d0, 3),
			tau_s, tau_h, disp_max);
	}
	checkCudaError(L);
	return 0;
}
#endif

__global__ void interpolate_mismatch(float *d0, float *outlier, float *out, int size, int dim2, int dim3)
{
	const float dir[] = {
		0	,  1,
		-0.5,  1,
		-1	,  1,
		-1	,  0.5,
		-1	,  0,
		-1	, -0.5,
		-1	, -1,
		-0.5, -1,
		0	, -1,
		0.5 , -1,
		1	, -1,
		1	, -0.5,
		1	,  0,
		1	,  0.5,
		1	,  1,
		0.5 ,  1
	};

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		if (outlier[id] != 2) {
			out[id] = d0[id];
			return;
		}

		float vals[16];
		int vals_size = 0;

		int x = id % dim3;
		int y = id / dim3;
		for (int d = 0; d < 16; d++) {
			float dx = dir[2 * d];
			float dy = dir[2 * d + 1];
			float xx = x;
			float yy = y;
			int xx_i = round(xx);
			int yy_i = round(yy);
			while (0 <= yy_i && yy_i < dim2 && 0 <= xx_i && xx_i < dim3 && outlier[yy_i * dim3 + xx_i] == 2) {
				xx += dx;
				yy += dy;
				xx_i = round(xx);
				yy_i = round(yy);
			}

			int ind = yy_i * dim3 + xx_i;
			if (0 <= yy_i && yy_i < dim2 && 0 <= xx_i && xx_i < dim3) {
				assert(outlier[ind] != 2);
				vals[vals_size++] = d0[ind];
			}
		}
		assert(vals_size > 0);
		sort(vals, vals_size);
		out[id] = vals[vals_size / 2];
	}
}

int interpolate_mismatch(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *d0 = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *outlier = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *out = new_tensor_like(state, d0);

	interpolate_mismatch<<<(THCudaTensor_nElement(state, out) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, d0),
		THCudaTensor_data(state, outlier),
		THCudaTensor_data(state, out),
		THCudaTensor_nElement(state, out),
		THCudaTensor_size(state, out, 2),
		THCudaTensor_size(state, out, 3));
	checkCudaError(L);
	luaT_pushudata(L, out, "torch.CudaTensor");
	return 1;
}

__global__ void interpolate_occlusion(float *d0, float *outlier, float *out, int size, int dim3)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		if (outlier[id] != 1) {
			out[id] = d0[id];
			return;
		}
		int x = id % dim3;

		int dx = 0;
		while (x + dx >= 0 && outlier[id + dx] != 0) {
			dx--;
		}
		if (x + dx < 0) {
			dx = 0;
			while (x + dx < dim3 && outlier[id + dx] != 0) {
				dx++;
			}
		}
		if (x + dx < dim3) {
			out[id] = d0[id + dx];
		} else {
			out[id] = d0[id];
		}
	}
}

int interpolate_occlusion(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *d0 = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *outlier = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *out = new_tensor_like(state, d0);

	interpolate_occlusion<<<(THCudaTensor_nElement(state, out) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, d0),
		THCudaTensor_data(state, outlier),
		THCudaTensor_data(state, out),
		THCudaTensor_nElement(state, out),
		THCudaTensor_size(state, out, 3)
	);

	checkCudaError(L);
	luaT_pushudata(L, out, "torch.CudaTensor");
	return 1;
}

#if 0

__global__ void sobel(float *x, float *g1, float *g2, int size, int dim2, int dim3)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int xx = id % dim3;
		int yy = id / dim3;

		if (1 <= yy && yy < dim2 - 1 && 1 <= xx && xx < dim3 - 1) {
			g1[id] = -x[id-dim3-1] +x[id-dim3+1] -2*x[id-1] +2*x[id+1] -x[id+dim3-1] +x[id+dim3+1];
			g2[id] = x[id-dim3-1] +2*x[id-dim3] +x[id-dim3+1] -x[id+dim3-1] -2*x[id+dim3] -x[id+dim3+1];
		} else {
			g1[id] = 0;
			g2[id] = 0;
		}
	}
}

int sobel(lua_State *L) {
	THCudaTensor *x = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *g1 = new_tensor_like(x);
	THCudaTensor *g2 = new_tensor_like(x);

	sobel<<<(THCudaTensor_nElement(x) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(x),
		THCudaTensor_data(g1),
		THCudaTensor_data(g2),
		THCudaTensor_nElement(x),
		THCudaTensor_size(x, 2),
		THCudaTensor_size(x, 3)
	);

	checkCudaError(L);
	luaT_pushudata(L, g1, "torch.CudaTensor");
	luaT_pushudata(L, g2, "torch.CudaTensor");
	return 2;
}

__global__ void depth_discontinuity_adjustment(float *d0, float *dg1, float *dg2, float *xg1, float *xg2, float *out, int size, int dim3, float tau_e)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		if (abs(dg1[id]) > tau_e) {
			out[id] = xg1[id - 1] > xg1[id + 1] ? d0[id - 1] : d0[id + 1];
		} else if (abs(dg2[id]) > tau_e) {
			out[id] = xg2[id - dim3] > xg2[id + dim3] ? d0[id - dim3] : d0[id + dim3];
		} else {
			out[id] = d0[id];
		}
	}
}

int depth_discontinuity_adjustment(lua_State *L) {
	THCudaTensor *d0 = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *dg1 = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *dg2 = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	THCudaTensor *xg1 = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
	THCudaTensor *xg2 = (THCudaTensor*)luaT_checkudata(L, 5, "torch.CudaTensor");
	float tau_e = luaL_checknumber(L, 6);
	THCudaTensor *out = new_tensor_like(d0);

	depth_discontinuity_adjustment<<<(THCudaTensor_nElement(out) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(d0),
		THCudaTensor_data(dg1),
		THCudaTensor_data(dg2),
		THCudaTensor_data(xg1),
		THCudaTensor_data(xg2),
		THCudaTensor_data(out),
		THCudaTensor_nElement(out),
		THCudaTensor_size(out, 3),
		tau_e);
	checkCudaError(L);
	luaT_pushudata(L, out, "torch.CudaTensor");
	return 1;
}
#endif

__global__ void subpixel_enchancement(float *d0, float *c2, float *out, int size, int dim23, int disp_max) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int d = d0[id];
		out[id] = d;
		if (1 <= d && d < disp_max - 1) {
			float cn = c2[(d - 1) * dim23 + id];
			float cz = c2[d * dim23 + id];
			float cp = c2[(d + 1) * dim23 + id];
			float denom = 2 * (cp + cn - 2 * cz);
			if (denom > 1e-5) {
				out[id] = d - min(1.0, max(-1.0, (cp - cn) / denom));
			}
		}
	}
}

int subpixel_enchancement(lua_State *L) {
	THCState *state = getCutorchState(L);
	THCudaTensor *d0 = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *c2 = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	int disp_max = luaL_checkinteger(L, 3);
	THCudaTensor *out = new_tensor_like(state, d0);

	subpixel_enchancement<<<(THCudaTensor_nElement(state, out) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, d0),
		THCudaTensor_data(state, c2),
		THCudaTensor_data(state, out),
		THCudaTensor_nElement(state, out),
		THCudaTensor_size(state, out, 2) * THCudaTensor_size(state, out, 3),
		disp_max);
	checkCudaError(L);
	luaT_pushudata(L, out, "torch.CudaTensor");
	return 1;
}

__global__ void mean2d(float *img, float *kernel, float *out, int size, int kernel_radius, int dim2, int dim3, float alpha2)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int x = id % dim3;
		int y = id / dim3;

		float sum = 0;
		float cnt = 0;
		int i = 0;
		for (int xx = x - kernel_radius; xx <= x + kernel_radius; xx++) {
			for (int yy = y - kernel_radius; yy <= y + kernel_radius; yy++, i++) {
				if (0 <= xx && xx < dim3 && 0 <= yy && yy < dim2 && abs(img[yy * dim3 + xx] - img[y * dim3 + x]) < alpha2) {
					sum += img[yy * dim3 + xx] * kernel[i];
					cnt += kernel[i];
				}
			}
		}
		out[id] = sum / cnt;
	}
}

int mean2d(lua_State *L) {
	THCState *state = getCutorchState(L);
	THCudaTensor *img = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *kernel = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	float alpha2 = luaL_checknumber(L, 3);
	THCudaTensor *out = new_tensor_like(state, img);
	assert(THCudaTensor_size(state, kernel, 0) % 2 == 1);
	mean2d<<<(THCudaTensor_nElement(state, out) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, img),
		THCudaTensor_data(state, kernel),
		THCudaTensor_data(state, out),
		THCudaTensor_nElement(state, out),
		THCudaTensor_size(state, kernel, 0) / 2,
		THCudaTensor_size(state, out, 2),
		THCudaTensor_size(state, out, 3),
		alpha2);
	checkCudaError(L);
	luaT_pushudata(L, out, "torch.CudaTensor");
	return 1;
}

__global__ void Normalize_get_norm_(float *input, float *norm, int size1, int size23, int size023)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size023) {
		int dim23 = id % size23;
		int dim0 = id / size23;

		float sum = 0.0;
		for (int dim1 = 0; dim1 < size1; dim1++) {
			float x = input[(dim0 * size1 + dim1) * size23 + dim23];
			sum += x * x;
		}
		norm[dim0 * size23 + dim23] = sum + 1e-5;
	}
}

__global__ void Normalize_forward_(float *input, float *norm, float *output, int size23, int size123, int size0123)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size0123) { 
		int dim23 = id % size23;
		int dim0 = (id / size123);
		output[id] = input[id] / sqrtf(norm[dim0 * size23 + dim23]);
	}
}

int Normalize_forward(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *norm = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *output = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");

	Normalize_get_norm_<<<(THCudaTensor_nElement(state, norm) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, input),
		THCudaTensor_data(state, norm),
		THCudaTensor_size(state, input, 1),
		THCudaTensor_size(state, input, 2) * THCudaTensor_size(state, input, 3),
		THCudaTensor_nElement(state, norm));

	Normalize_forward_<<<(THCudaTensor_nElement(state, output) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, input),
		THCudaTensor_data(state, norm),
		THCudaTensor_data(state, output),
		THCudaTensor_size(state, input, 2) * THCudaTensor_size(state, input, 3),
		THCudaTensor_size(state, input, 1) * THCudaTensor_size(state, input, 2) * THCudaTensor_size(state, input, 3),
		THCudaTensor_nElement(state, output));
	checkCudaError(L);
	return 0;
}

__global__ void Normalize_backward_input_(float *grad_output, float *input, float *norm, float *grad_input, int size1, int size23, int size0123)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size0123) {
		int dim0 = id;
		int dim23 = dim0 % size23;
		dim0 /= size23;
		int dim1 = dim0 % size1;
		dim0 /= size1;

		float denom = powf(norm[dim0 * size23 + dim23], 1.5);
		float deriv = (norm[dim0 * size23 + dim23] - input[id] * input[id]) / denom * grad_output[id];

		float sum = 0;
		for (int dim1_ = 0; dim1_ < size1; dim1_++) {
			if (dim1_ != dim1) {
				int ind = (dim0 * size1 + dim1_) * size23 + dim23;
				sum += input[ind] * grad_output[ind];
			}
		}
		grad_input[id] = deriv - sum * input[id] / denom;
	}
}

int Normalize_backward_input(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *grad_output = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *norm = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	THCudaTensor *grad_input = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");

	Normalize_backward_input_<<<(THCudaTensor_nElement(state, input) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, grad_output),
		THCudaTensor_data(state, input),
		THCudaTensor_data(state, norm),
		THCudaTensor_data(state, grad_input),
		THCudaTensor_size(state, input, 1),
		THCudaTensor_size(state, input, 2) * THCudaTensor_size(state, input, 3),
		THCudaTensor_nElement(state, input));
	checkCudaError(L);
	return 0;
}

struct Margin2_functor {
	float margin;
	__host__ Margin2_functor(float margin_) : margin(margin_) {};
	__device__ float forward(float pos, float neg) {
		return fmaxf(0, neg - pos + margin);
	}
	__device__ float backward(float pos, float neg, int which) {
		float f = neg - pos + margin;
		if (which == 0) {
			return -1. * (f > 0);
		} else {
			return f > 0;
		}
	}
};

struct Margin2_squared_functor {
	float margin;
	__host__ Margin2_squared_functor(float margin_) : margin(margin_) {};
	__device__ float forward(float pos, float neg) {
		float d = fmaxf(0, neg - pos + margin);
		return d * d * 0.5;
	}
	__device__ float backward(float pos, float neg, int which) {
		float f = neg - pos + margin;
		if (which == 0) {
			return -f * (f > 0);
		} else {
			return f * (f > 0);
		}
	}
};

template <class Op>
__global__ void Margin2_(float *input, float *tmp, float *gradInput, float margin, Op op, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		float pos = input[id * 2];
		float neg = input[id * 2 + 1];
		tmp[id] = op.forward(pos, neg);
		gradInput[id * 2] = op.backward(pos, neg, 0);
		gradInput[id * 2 + 1] = op.backward(pos, neg, 1);
	}
}

int Margin2(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *tmp = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *gradInput = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	float margin = luaL_checknumber(L, 4);
	int pow = luaL_checkinteger(L, 5);

	if (pow == 1) {
		Margin2_<<<(THCudaTensor_nElement(state, tmp) - 1) / TB + 1, TB>>>(
			THCudaTensor_data(state, input),
			THCudaTensor_data(state, tmp),
			THCudaTensor_data(state, gradInput),
			margin,
			Margin2_functor(margin),
			THCudaTensor_nElement(state, tmp));
	} else if (pow == 2) {
		Margin2_<<<(THCudaTensor_nElement(state, tmp) - 1) / TB + 1, TB>>>(
			THCudaTensor_data(state, input),
			THCudaTensor_data(state, tmp),
			THCudaTensor_data(state, gradInput),
			margin,
			Margin2_squared_functor(margin),
			THCudaTensor_nElement(state, tmp));
	}
	checkCudaError(L);
	return 0;
}

__global__ void StereoJoin_(float *input_L, float *input_R, float *output_L, float *output_R, int size1_input, int size1, int size3, int size23)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size23) {
		int dim3 = id % size3;
		assert(size1_input <= 128);
		float L_cache[128];
		for (int i = 0; i < size1_input; i++) {
			L_cache[i] = input_L[i * size23 + id];
		}

		for (int d = 0; d < size1; d++) {
			if (dim3 - d >= 0) {
				float sum = 0;
				for (int i = 0; i < size1_input; i++) {
					sum -= L_cache[i] * input_R[i * size23 + id - d];
				}
				output_L[d * size23 + id] = sum;
				output_R[d * size23 + id - d] = sum;
			}
		}
	}
}

int StereoJoin(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *input_L = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *input_R = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	THCudaTensor *output_L = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
	THCudaTensor *output_R = (THCudaTensor*)luaT_checkudata(L, 4, "torch.CudaTensor");
	int size23 = THCudaTensor_size(state, output_L, 2) * THCudaTensor_size(state, output_L, 3);
	StereoJoin_<<<(size23 - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, input_L),
		THCudaTensor_data(state, input_R),
		THCudaTensor_data(state, output_L),
		THCudaTensor_data(state, output_R),
		THCudaTensor_size(state, input_L, 1),
		THCudaTensor_size(state, output_L, 1),
		THCudaTensor_size(state, output_L, 3),
		size23);
	checkCudaError(L);
	return 0;
}

__global__ void StereoL2R_(float *vol_L, float *vol_R, int size2, int size3, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int dim3 = id % size3;
		int dim1 = id / (size2 * size3);

		if (dim3 + dim1 >= size3) {
			vol_R[id] = HIP_INF;
		} else {
			vol_R[id] = vol_L[id + dim1];
		}
	}
}

int StereoL2R(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *vol_L = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *vol_R = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
	StereoL2R_<<<(THCudaTensor_nElement(state, vol_L) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, vol_L),
		THCudaTensor_data(state, vol_R),
		THCudaTensor_size(state, vol_R, 2),
		THCudaTensor_size(state, vol_R, 3),
		THCudaTensor_nElement(state, vol_R));
	checkCudaError(L);
	return 0;
}

__global__ void bilateral_filter(float *img, float *out, int size, int dim2, int dim3, int kernel_radius, float sigma1, float sigma2)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int x = id % dim3;
		int y = id / dim3;

		float sum = 0;
		float cnt = 0;
		for (int i = -kernel_radius; i <= kernel_radius; i++) {
			for (int j = -kernel_radius; j <= kernel_radius; j++) {
				int yy = y + i;
				int xx = x + j;
				if (0 <= xx && xx < dim3 && 0 <= yy && yy < dim2) {
					float color_diff = img[yy * dim3 + xx] - img[y * dim3 + x];
					float v1 = exp(-(i * i + j * j) / (2 * sigma1 * sigma1));
					float v2 = exp(-(color_diff * color_diff) / (2 * sigma2 * sigma2));
					sum += img[yy * dim3 + xx] * v1 * v2;
					cnt += v1 * v2;
				}
			}
		}
		out[id] = sum / cnt;
	}
}

int bilateral_filter(lua_State *L) {
	THCState *state = getCutorchState(L);
	THCudaTensor *img = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	float sigma1 = luaL_checknumber(L, 2);
	float sigma2 = luaL_checknumber(L, 3);
	THCudaTensor *out = new_tensor_like(state, img);
	int kernel_radius = ceil(min(sigma1, sigma2) * 3);
	bilateral_filter<<<(THCudaTensor_nElement(state, out) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, img),
		THCudaTensor_data(state, out),
		THCudaTensor_nElement(state, out),
		THCudaTensor_size(state, out, 2),
		THCudaTensor_size(state, out, 3),
		kernel_radius, sigma1, sigma2);
	checkCudaError(L);
	luaT_pushudata(L, out, "torch.CudaTensor");
	return 1;
}

__global__ void median2d(float *img, float *out, int size, int dim2, int dim3, int kernel_radius)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int x = id % dim3;
		int y = id / dim3;

		float xs[11 * 11];
		int xs_size = 0;
		for (int xx = x - kernel_radius; xx <= x + kernel_radius; xx++) {
			for (int yy = y - kernel_radius; yy <= y + kernel_radius; yy++) {
				if (0 <= xx && xx < dim3 && 0 <= yy && yy < dim2) {
					xs[xs_size++] = img[yy * dim3 + xx];
				}
			}
		}
		sort(xs, xs_size);
		out[id] = xs[xs_size / 2];
	}
}

int median2d(lua_State *L) {
	THCState *state = getCutorchState(L);
	THCudaTensor *img = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	int kernel_size = luaL_checkinteger(L, 2);
	THCudaTensor *out = new_tensor_like(state, img);
	assert(kernel_size % 2 == 1);
	assert(kernel_size <= 11);
	median2d<<<(THCudaTensor_nElement(state, out) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, img),
		THCudaTensor_data(state, out),
		THCudaTensor_nElement(state, out),
		THCudaTensor_size(state, out, 2),
		THCudaTensor_size(state, out, 3),
		kernel_size / 2);
	checkCudaError(L);
	luaT_pushudata(L, out, "torch.CudaTensor");
	return 1;
}

#if 0
int histogram(lua_State *L) {
	THFloatTensor *img = (THFloatTensor*)luaT_checkudata(L, 1, "torch.FloatTensor");
	THIntTensor *hist = THIntTensor_newWithSize1d(256);
	THIntTensor_zero(hist);

	float *img_data = THFloatTensor_data(img);
	int *hist_data = THIntTensor_data(hist);
	for (int i = 0; i < THFloatTensor_size(img, 2) * THFloatTensor_size(img, 3); i++) {
		assert(0 <= img_data[i] && img_data[i] < 256);
		hist_data[(int)img_data[i]]++;
	}
	luaT_pushudata(L, hist, "torch.IntTensor");
	return 1;
}

int histogram_equalization_map(lua_State *L) {
	THIntTensor *cdf = (THIntTensor*)luaT_checkudata(L, 1, "torch.IntTensor");
	THIntTensor *map = THIntTensor_new();
	THIntTensor_resizeAs(map, cdf);

	int *cdf_data = THIntTensor_data(cdf);
	int max = cdf_data[255];
	int min = cdf_data[0];
	for (int i = 0; i < 256; i++) {
		if (cdf_data[i]) {
			min = cdf_data[i];
			break;
		}
	}
	int *map_data = THIntTensor_data(map);
	for (int i = 0; i < 256; i++) {
		map_data[i] = round((double)(cdf_data[i] - min) / (max - min) * 255);
	}
	luaT_pushudata(L, map, "torch.IntTensor");
	return 1;
}

int map_intensities(lua_State *L) {
	THFloatTensor *img = (THFloatTensor*)luaT_checkudata(L, 1, "torch.FloatTensor");
	THIntTensor *map = (THIntTensor*)luaT_checkudata(L, 2, "torch.IntTensor");
	THFloatTensor *out = THFloatTensor_new();
	THFloatTensor_resizeAs(out, img);

	float *img_data = THFloatTensor_data(img);
	float *out_data = THFloatTensor_data(out);
	int *map_data = THIntTensor_data(map);
	for (int i = 0; i < THFloatTensor_size(img, 2) * THFloatTensor_size(img, 3); i++) {
		out_data[i] = map_data[(int)img_data[i]];
	}
	luaT_pushudata(L, out, "torch.FloatTensor");
	return 1;
}
#endif

int readPNG16(lua_State *L)
{
	THFloatTensor *img_ = (THFloatTensor*)luaT_checkudata(L, 1, "torch.FloatTensor");
	const char* fname = luaL_checkstring(L, 2);

	float *img = THFloatTensor_data(img_);		
	png::image<png::gray_pixel_16> image(fname);
	int width = image.get_width();
	int height = image.get_height();
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			uint16_t val = image.get_pixel(j, i);
			img[i * width + j] = val == 0 ? 0.0 : ((float)val)/256.0;
		}
	}
	return 0;
}

int writePNG16(lua_State *L)
{
	THFloatTensor *img_ = (THFloatTensor*)luaT_checkudata(L, 1, "torch.FloatTensor");
	int height = luaL_checkinteger(L, 2);
	int width = luaL_checkinteger(L, 3);
	const char* fname = luaL_checkstring(L, 4);

	float *img = THFloatTensor_data(img_);		
	png::image<png::gray_pixel_16> image(width, height);
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			float val = img[i * width + j];			
			image.set_pixel(j, i, (uint16_t)(val < 1e-5 ? 0 : val * 256));
		}
	}
	image.write(fname);
	return 0;
}

int writePFM(lua_State *L)
{
	THFloatTensor *img_ = (THFloatTensor*)luaT_checkudata(L, 1, "torch.FloatTensor");
	const char* fname = luaL_checkstring(L, 2);

	int height = THFloatTensor_size(img_, 0);
	int width = THFloatTensor_size(img_, 1);

	FILE *f = fopen(fname, "w");
	fprintf(f, "Pf\n%d %d\n-0.003922\n", width, height);
	fwrite(THFloatTensor_data(img_), 4, height * width, f);
	fclose(f);

	return 0;
}

__global__ void remove_nonvisible(float *y, int size, int size3)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int x = id % size3;
		if (y[id] >= x) {
			y[id] = 0;
		}
	}
}

int remove_nonvisible(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *y = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");

	remove_nonvisible<<<(THCudaTensor_nElement(state, y) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, y),
		THCudaTensor_nElement(state, y),
		THCudaTensor_size(state, y, 3));
	checkCudaError(L);
	return 0;
}

__global__ void remove_occluded(float *y, int size, int size3)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int x = id % size3;
		for (int i = 1; x + i < size3; i++) {
			if (i - y[id + i] < -y[id]) {
				y[id] = 0;
				break;
			}
		}
	}
}

int remove_occluded(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *y = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");

	remove_occluded<<<(THCudaTensor_nElement(state, y) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, y), 
		THCudaTensor_nElement(state, y),
		THCudaTensor_size(state, y, 3));
	checkCudaError(L);
	return 0;
}

__global__ void remove_white(float *x, float *y, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		if (x[id] == 255) {
			y[id] = 0;
		}
	}
}

int remove_white(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *x = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *y = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");

	remove_white<<<(THCudaTensor_nElement(state, y) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, x),
		THCudaTensor_data(state, y),
		THCudaTensor_nElement(state, y));
	checkCudaError(L);
	return 0;
}

__global__ void copy_fill(float *in, float *out, int size, int in_size2, int in_size3, int out_size2, int out_size3)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < size) {
		int out_x = id % out_size3;
		int out_y = id / out_size3;

		int in_x = out_x - (out_size3 - in_size3) / 2;
		int in_y = out_y - (out_size2 - in_size2) / 2;

		int x = min(in_size3 - 1, max(0, in_x));
		int y = min(in_size2 - 1, max(0, in_y));

		out[id] = in[y * in_size3 + x];
	}
}

int copy_fill(lua_State *L)
{
	THCState *state = getCutorchState(L);
	THCudaTensor *in = (THCudaTensor*)luaT_checkudata(L, 1, "torch.CudaTensor");
	THCudaTensor *out = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");

	copy_fill<<<(THCudaTensor_nElement(state, out) - 1) / TB + 1, TB>>>(
		THCudaTensor_data(state, in),
		THCudaTensor_data(state, out),
		THCudaTensor_nElement(state, out),
		THCudaTensor_size(state, in, 2),
		THCudaTensor_size(state, in, 3),
		THCudaTensor_size(state, out, 2),
		THCudaTensor_size(state, out, 3));
	checkCudaError(L);
	luaT_pushudata(L, out, "torch.CudaTensor");
	return 1;
}

void memcpy2d(float *dst, float *src, int x, int y, int win_radius, int height, int width)
{
	assert(0 <= x - win_radius);
	assert(x + win_radius <= width);
	assert(0 <= y - win_radius);
	assert(y + win_radius <= height);
	for (int i = -win_radius; i <= win_radius; i++) {
		memcpy(dst, src + (y + i) * width + x - win_radius, (win_radius * 2 + 1) * sizeof(float));
		dst += win_radius * 2 + 1;
	}
}

double random_uniform() 
{
	return ((double)rand()/(double)RAND_MAX);
}

int random_int(int a, int b)
{
	assert(a <= b);
	return floor(random_uniform() * (b - a + 1) + a);
}

double random_exp(double lambda) 
{
	double u = random_uniform();
	return -log(u) / lambda;
}

int subset_dataset(lua_State *L)
{
	THLongTensor *index_ = (THLongTensor*)luaT_checkudata(L, 1, "torch.LongTensor");
	THFloatTensor *input_ = (THFloatTensor*)luaT_checkudata(L, 2, "torch.FloatTensor");
	THFloatTensor *output_ = (THFloatTensor*)luaT_checkudata(L, 3, "torch.FloatTensor");

	long *index = THLongTensor_data(index_);
	float *input = THFloatTensor_data(input_);
	float *output = THFloatTensor_data(output_);

	const int N = 200;

	int set[N];
	for (int i = 0; i < N; i++) {
		set[i] = 0;
	}

	for (int i = 0; i < THLongTensor_nElement(index_); i++) {
		assert(index[i] < N);
		set[index[i]] = 1;
	}

	int i = 0;
	for (int j = 0; j < THFloatTensor_size(input_, 0); j++) {
		int im = input[j * 4];
		if (set[im]) {
			for (int k = 0; k < 4; k++) {
				output[i * 4 + k] = input[j * 4 + k];
			}
			i++;
		}
	}

	lua_pushinteger(L, i);
	return 1;
}

int make_dataset2(lua_State *L)
{
	THFloatTensor *disp_ = (THFloatTensor*)luaT_checkudata(L, 1, "torch.FloatTensor");
	THFloatTensor *nnz_ = (THFloatTensor*)luaT_checkudata(L, 2, "torch.FloatTensor");
	int img = luaL_checkinteger(L, 3);
	int t = luaL_checkinteger(L, 4);

	float *disp = THFloatTensor_data(disp_);
	float *nnz = THFloatTensor_data(nnz_);

	int height = THFloatTensor_size(disp_, 2);
	int width = THFloatTensor_size(disp_, 3);

	
	int nnz_size = THFloatTensor_nElement(nnz_);

	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			if (disp[i * width + j] > 0.5) {
				assert(t * 4 + 4 <= nnz_size);
				nnz[t * 4 + 0] = img; //id
				nnz[t * 4 + 1] = i; // y
				nnz[t * 4 + 2] = j; // x
				nnz[t * 4 + 3] = disp[i * width + j]; //disparity
				t++;
			}
		}
	}

	lua_pushinteger(L, t);
	return 1;
}

int make_dataset(lua_State *L)
{
	THFloatTensor *x0_ = (THFloatTensor*)luaT_checkudata(L, 1, "torch.FloatTensor");
	THFloatTensor *x1_ = (THFloatTensor*)luaT_checkudata(L, 2, "torch.FloatTensor");
	THFloatTensor *disp_ = (THFloatTensor*)luaT_checkudata(L, 3, "torch.FloatTensor");
	THFloatTensor *x_ = (THFloatTensor*)luaT_checkudata(L, 4, "torch.FloatTensor");
	THFloatTensor *y_ = (THFloatTensor*)luaT_checkudata(L, 5, "torch.FloatTensor");
	int t = luaL_checkinteger(L, 6);
	float thr_true = luaL_checknumber(L, 7);
	float thr_false_l = luaL_checknumber(L, 8);
	float thr_false_u = luaL_checknumber(L, 9);

	float *x0 = THFloatTensor_data(x0_);
	float *x1 = THFloatTensor_data(x1_);
	float *disp = THFloatTensor_data(disp_);
	float *x = THFloatTensor_data(x_);
	float *y = THFloatTensor_data(y_);

	int height = THFloatTensor_size(x0_, 2);
	int width = THFloatTensor_size(x0_, 3);
	int win_size = THFloatTensor_size(x_, 2);
	int x_size = THFloatTensor_size(x_, 0);
	assert(win_size % 2 == 1);
	int win_radius = (win_size - 1) / 2;
	
	x += t * 2 * win_size * win_size;
	for (int i = win_radius; i < height - win_radius; i++) {
		for (int j = win_radius; j < width - win_radius; j++) {
			if (disp[i * width + j] > 0.5) {
				int d_true = round(disp[i * width + j]);
				if (0 <= j - d_true - win_radius) {
					/* true offset */
					int delta = 0;
					for (;;) {
						delta = random_int(-thr_true, thr_true);
						if (0 <= j - d_true + delta - win_radius && j - d_true + delta + win_radius < width) {
							break;
						}
					}
					assert(t < x_size);
					memcpy2d(x, x0, j, i, win_radius, height, width); x += win_size * win_size;
					memcpy2d(x, x1, j - d_true + delta, i, win_radius, height, width); x += win_size * win_size;
					y[t] = 1;
					t++;

					/* false offset */
					delta = 0;
					for (;;) {
						delta = random_int(thr_false_l, thr_false_u);
						if (random_uniform() < 0.5) {
							delta = -delta;
						}
						if (0 <= j - d_true + delta - win_radius && j - d_true + delta + win_radius < width) {
							break;
						}
					}
					assert(t < x_size);
					memcpy2d(x, x0, j, i, win_radius, height, width); x += win_size * win_size;
					memcpy2d(x, x1, j - d_true + delta, i, win_radius, height, width); x += win_size * win_size;
					y[t] = 0;
					t++;
				}
			}
		}
	}
	lua_pushinteger(L, t);
	return 1;
}

/* CPU implementation */
int grey2jet(lua_State *L)
{
	THDoubleTensor *grey_img = (THDoubleTensor*)luaT_checkudata(L, 1, "torch.DoubleTensor");
	THDoubleTensor *col_img = (THDoubleTensor*)luaT_checkudata(L, 2, "torch.DoubleTensor");

	assert(grey_img->nDimension == 2);
	if (3 * THDoubleTensor_nElement(grey_img) != THDoubleTensor_nElement(col_img)) {
		luaL_error(L, "Size mismatch");
	}

	int height = THDoubleTensor_size(grey_img, 0);
	int width = THDoubleTensor_size(grey_img, 1);

	double *gray_data = THDoubleTensor_data(grey_img);
	double *col_data = THDoubleTensor_data(col_img);

	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			double val = gray_data[i * width + j] * 4;
			double r = 0, g = 0, b = 0;

			if (-0.1 <= val && val < 0.5) {
				r = 0;
				g = 0;
				b = 0.5 + val;
			} else if (0.5 <= val && val < 1.5) {
				r = 0;
				g = val - 0.5;
				b = 1;
			} else if (1.5 <= val && val < 2.5) {
				r = val - 1.5;
				g = 1;
				b = 1 - (val - 1.5);
			} else if (2.5 <= val && val < 3.5) {
				r = 1;
				g = 1 - (val - 2.5);
				b = 0;
			} else if (3.5 <= val && val <= 4.1) {
				r = 1 - (val - 3.5);
				g = 0;
				b = 0;
			} else {
				printf("val = %f\n", val);
				assert(0);
			}

			col_data[(0 * height + i) * width + j] = r;
			col_data[(1 * height + i) * width + j] = g;
			col_data[(2 * height + i) * width + j] = b;
		}
	}
	return 0;
}

int version(lua_State* L)
{
	printf("libadcensus version 0.0.5\n");
	return 0;
}

static const struct luaL_Reg funcs[] = {
	{"ad", ad},
	{"census", census},
	{"cross", cross},
	{"cbca", cbca},
	{"sgm", sgm},
	{"sgm2", sgm2},
	{"sgm3", sgm3},
	{"outlier_detection", outlier_detection},
	{"interpolate_occlusion", interpolate_occlusion},
	{"interpolate_mismatch", interpolate_mismatch},
	{"subpixel_enchancement", subpixel_enchancement},
	{"copy_fill", copy_fill},
	{"median2d", median2d},
	{"mean2d", mean2d},
	{"Normalize_forward", Normalize_forward},
	{"Normalize_backward_input", Normalize_backward_input},
	{"Margin2", Margin2},
	{"StereoJoin", StereoJoin},
	{"StereoL2R", StereoL2R},

	{"subset_dataset", subset_dataset},
	{"make_dataset", make_dataset},
	{"make_dataset2", make_dataset2},
	{"remove_nonvisible", remove_nonvisible},
	{"remove_occluded", remove_occluded},
	{"remove_white", remove_white},
	{"readPNG16", readPNG16},
	{"writePNG16", writePNG16},
	{"writePFM", writePFM},
	{"grey2jet", grey2jet},
	{"spatial_argmin", spatial_argmin},

	{"version", version},
	{NULL, NULL}
};

#include "SpatialLogSoftMax.cu"

extern "C" int luaopen_libadcensus(lua_State *L) {
	srand(42);
	cunn_SpatialLogSoftMax_init(L);
	luaL_openlib(L, "adcensus", funcs, 0);
	return 1;
}
